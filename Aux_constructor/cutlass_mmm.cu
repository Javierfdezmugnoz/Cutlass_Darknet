#include <iostream>
#include <cutlass/numeric_types.h>
#include <cutlass/core_io.h>
#include <cutlass/cutlass.h>
#include <cutlass/gemm/device/gemm.h>
// #include <MMM_protected.h>
#include "hip/hip_runtime.h"

//Define a CUTLASS GEMM template and launch a GEMM kernel.
//hipError_t CutlassSgemmNN(int M, int N, int K, float alpha, float const *A, int lda, float const *B, int ldb, float beta, float *C, int ldc, uint32_t *d_ES_a, uint32_t *d_ES_b, uint32_t *d_ES_c) 
hipError_t CutlassSgemmNN(int M, int N, int K, float alpha, float *A, int lda, float *B, int ldb, float beta, float *C, int ldc, uint32_t *d_ES_a, uint32_t *d_ES_b, uint32_t *d_ES_c) 
{
  // Define type definition for single-precision CUTLASS GEMM with column-major
  // input matrices and 128x128x8 threadblock tile size (chosen by default).
  //
  // To keep the interface manageable, several helpers are defined for plausible compositions
  // including the following example for single-precision GEMM. Typical values are used as
  // default template arguments. See `cutlass/gemm/device/default_gemm_configuration.h` for more details.
  //
  // To view the full gemm device API interface, see `cutlass/gemm/device/gemm.h`

   using ColumnMajor = cutlass::layout::ColumnMajor;
  // std::cout << M << "\t N:" << N << "\t K:" << K << "\t A:" << A[0] << "\t B:" << B[0] << "\t C:" << C[0] << "\t alpha:" << alpha << "\t beta:" << beta << "\t" << "d_ES_a[0]" << "\t" << "d_ES_b[0]" << "\t" <<" d_ES_c[0]" << "\n" ;

  using CutlassGemm = cutlass::gemm::device::Gemm<float,        // Data-type of A matrix
                                                  ColumnMajor,  // Layout of A matrix
                                                  float,        // Data-type of B matrix
                                                  ColumnMajor,  // Layout of B matrix
                                                  float,        // Data-type of C matrix
                                                  ColumnMajor>; // Layout of C matrix

  // // Define a CUTLASS GEMM type
   CutlassGemm gemm_operator;

  //printf("\n Direction of h_ES_0: %p and value: %f \n", (void *) h_ES_0, h_ES_0[4]);

  // Construct the CUTLASS GEMM arguments object.
  //
  // One of CUTLASS's design patterns is to define gemm argument objects that are constructible
  // in host code and passed to kernels by value. These may include pointers, strides, scalars,
  // and other arguments needed by Gemm and its components.
  //
  // The benefits of this pattern are (1.) a structured, composable strategy for passing host-constructible
  // arguments to kernels and (2.) minimized initialization overhead on kernel entry.
  //
  CutlassGemm::Arguments args({M , N, K},  // Gemm Problem dimensions
                              {A, lda},    // Tensor-ref for source matrix A
                              {B, ldb},    // Tensor-ref for source IImatrix B
                              {C, ldc},    // Tensor-ref for source matrix C
                              {C, ldc},    // Tensor-ref for destination matrix D (may be different memory than source C matrix)
                              {alpha, beta}// Scalars used in the Epilogue
                              ,d_ES_a,      // Pointer to d_ES_a
                              d_ES_b,      // Pointer to d_ES_b
                              d_ES_c
                              );     // Pointer to d_ES_c
                             

  // // Code included by JFdez: I have to include in args variable this: d_ES_0 and d_ES_1

  // //
  // // Launch the CUTLASS GEMM kernel.
  // // 
  // std::cout << "Here we are!\n";
  // std::cout << M << "\t N:" << N << "\t K:" << K << "\t A:" << A[0] << "\t B:" << B[0] << "\t C:" << C[0] << "\t alpha:" << alpha << "\t beta:" << beta << "\t" << "d_ES_a[0]" << "\t" << "d_ES_b[0]" << "\t" <<" d_ES_c[0]" << "\n" ;
  cutlass::Status status = gemm_operator(args);
  // std::cout << "Here we are?\n";
  // //
  // // Return a hipError_t if the CUTLASS GEMM operator returned an error code.
  // //

  if (status != cutlass::Status::kSuccess) {
    return hipErrorUnknown;
  }

  // Return success, if no errors were encountered.
  return hipSuccess;
}
